#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/random/uniform_int_distribution.h>
#include <thrust/random/linear_congruential_engine.h>
#include "global_structs.h"

using thrust::random::minstd_rand0;
using thrust::random::uniform_int_distribution;

//Create an random int array repesenting a solution to a TSP. For inisziation.
__global__ void createRandomPermutation(deviceFields fields){
	short tempResult[CHROMOSOME_SIZE];
	short temp;
	short rand;
	short * chromosome = fields.population[threadIdx.x+blockIdx.x*blockDim.x].chromosome;
//	short start = (threadIdx.x + blockIdx.x*blockDim.x)*CHROMOSOME_SIZE;

	minstd_rand0 rng(fields.seeds[threadIdx.x]);

	for(short i = 0; i < CHROMOSOME_SIZE; i++){
		tempResult[i] = fields.source[i];
	}

	for(short i = CHROMOSOME_SIZE-1; i > 0; i--){
		uniform_int_distribution<short> dist(0,i+1);
		rand = dist(rng);
		temp = tempResult[rand];
		tempResult[rand] = tempResult[i];
		tempResult[i] = temp;
	}
	__syncthreads();

	for(short i = 0; i < CHROMOSOME_SIZE; i++){
		chromosome[i] = tempResult[i];
	}
}

__global__ void createRandomSeeds(deviceFields fields, long seed){
	minstd_rand0 rng(seed*(threadIdx.x + blockIdx.x*blockDim.x)-34156);

	uniform_int_distribution<int> dist(0,RAND_MAX);
	fields.seeds[threadIdx.x + blockDim.x*blockIdx.x]=dist(rng);
}
