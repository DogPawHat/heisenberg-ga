#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/random/uniform_int_distribution.h>
#include <thrust/random/linear_congruential_engine.h>
#include "global_structs.h"

using thrust::random::minstd_rand0;
using thrust::random::uniform_int_distribution;

//Create an random int array repesenting a solution to a TSP. For inisziation.
__global__ void createRandomPermutation(deviceFields fields, long seed){
	short tempResult[CHROMOSOME_SIZE];
	short temp;
	short rand;
	short start = (threadIdx.x + blockIdx.x*blockDim.x)*CHROMOSOME_SIZE;

	minstd_rand0 rng(seed*(threadIdx.x + blockIdx.x*blockDim.x)-341256);

	for(short i = 0; i < CHROMOSOME_SIZE; i++){
		tempResult[i] = fields.source[i];
	}

	for(short i = CHROMOSOME_SIZE-1; i > 0; i--){
		uniform_int_distribution<short> dist(0,i+1);
		rand = dist(rng);
		temp = tempResult[rand];
		tempResult[rand] = tempResult[i];
		tempResult[i] = temp;
	}

	for(short i = 0; i < CHROMOSOME_SIZE; i++){
		fields.population[start+i] = tempResult[i];
	}
}

__global__ void createRandomSeeds(deviceFields fields, long seed){
	minstd_rand0 rng(seed*(threadIdx.x + blockIdx.x*blockDim.x)-34156);

	uniform_int_distribution<int> dist(0,RAND_MAX);
	fields.seeds[threadIdx.x + blockDim.x*blockIdx.x]=dist(rng);
}