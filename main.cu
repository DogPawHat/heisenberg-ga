#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include "global_structs.h"
#include "berlin52.h"

__global__ void createRandomPermutation(deviceFields fields);
__global__ void runGeneticAlgorithm(deviceFields fields);
__global__ void createRandomSeeds(deviceFields fields, long seed);

hipError_t check(hipError_t call){
	if(call != hipSuccess){
		throw call;
	}
}

int main(){
	try{
		deviceFields device;
		hostFields host;

		hipMalloc((void**) &device.population, POPULATION_SIZE*sizeof(metaChromosome));
		hipMalloc((void**) &device.seeds, POPULATION_SIZE*sizeof(int));
		hipMalloc((void**) &device.TSPGraph, 2*CHROMOSOME_SIZE*sizeof(float));
		hipMalloc((void**) &device.source, CHROMOSOME_SIZE*sizeof(short));

		hipMemcpy(device.TSPGraph, berlin52, 2*CHROMOSOME_SIZE*sizeof(float), hipMemcpyHostToDevice);

		check(hipDeviceSynchronize());
	
		thrust::device_ptr<short> sourceThrust(device.source);
		thrust::sequence(sourceThrust, sourceThrust+CHROMOSOME_SIZE);

		createRandomSeeds<<<GRID_SIZE, BLOCK_SIZE>>>(device, time(NULL));
		createRandomPermutation<<<GRID_SIZE, BLOCK_SIZE>>>(device);
		check(hipDeviceSynchronize());

		runGeneticAlgorithm<<<GRID_SIZE, BLOCK_SIZE>>>(device);
		check(hipDeviceSynchronize());

		hipMemcpy(host.population, device.population, POPULATION_SIZE*sizeof(metaChromosome),hipMemcpyDeviceToHost);

		for (short i = 0; i < POPULATION_SIZE; i++){
			for(short j = 0; j < CHROMOSOME_SIZE; j++){
				std::cout << host.population[i].chromosome[j] << " ";
			}
			std::cout << host.population[i].distance << " " << host.population[i].fitness << std::endl;
		}

		hipFree(device.population);
		check(hipDeviceSynchronize());

		std::cin.get();
	}catch(hipError_t e){
		std::cout << "Oh crap: " << e << std::endl;
		std::cin.get();
	}
}
