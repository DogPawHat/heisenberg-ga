#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include "global_structs.h"
#include "berlin52.h"

__global__ void createRandomPermutation(deviceFields fields);
__global__ void runGeneticAlgorithm(deviceFields fields);
__global__ void createRandomSeeds(deviceFields fields, long seed);

void check(hipError_t call){
	if(call != hipSuccess){
		throw &call;
	}
}

int chromosomeCheck(short chromosome[]){
	int k;
	for(int i = 0; i < CHROMOSOME_SIZE; i++){
		k = 0;
		for(int j = 0; j < CHROMOSOME_SIZE; j++){
			if(chromosome[j] == i){
				k++;
			}else if(chromosome[j] > CHROMOSOME_SIZE || chromosome[j] < 0){
				return 1;
			}
		}
		if(k != 1){
			return 1;
		}
	}
	return 0;
}

int main(){
	try{
		deviceFields device;
		hostFields host;

		hipMalloc((void**) &device.population, POPULATION_SIZE*sizeof(metaChromosome));
		hipMalloc((void**) &device.seeds, POPULATION_SIZE*sizeof(int));
		hipMalloc((void**) &device.TSPGraph, 2*CHROMOSOME_SIZE*sizeof(float));
		hipMalloc((void**) &device.source, CHROMOSOME_SIZE*sizeof(short));

		hipMemcpy(device.TSPGraph, berlin52, 2*CHROMOSOME_SIZE*sizeof(float), hipMemcpyHostToDevice);

		for(int i = 0; i < CHROMOSOME_SIZE; i++){
			host.source[i] = i;
		}

		hipMemcpy(device.source, host.source, CHROMOSOME_SIZE*sizeof(short), hipMemcpyHostToDevice);
		check(hipDeviceSynchronize());

		createRandomSeeds<<<GRID_SIZE, BLOCK_SIZE>>>(device, time(NULL));
		createRandomPermutation<<<GRID_SIZE, BLOCK_SIZE>>>(device);
		check(hipDeviceSynchronize());

		runGeneticAlgorithm<<<GRID_SIZE, BLOCK_SIZE>>>(device);
		check(hipDeviceSynchronize());

		hipMemcpy(host.population, device.population, POPULATION_SIZE*sizeof(metaChromosome),hipMemcpyDeviceToHost);

		for (short i = 0; i < POPULATION_SIZE; i++){
			std::cout << '[' << chromosomeCheck(host.population[i].chromosome) << ']' << " ";
			for(short j = 0; j < CHROMOSOME_SIZE; j++){
				std::cout << host.population[i].chromosome[j] << " ";
			}
			std::cout << /*host.population[i].distance <<*/ std::endl;
		}




		check(hipFree(device.population));
	}catch(hipError_t * e){
		std::cout << "Oh crap: " << *e << std::endl;
	}
}
