#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <vector>
#include <iostream>
#include <fstream>
#include "rand.h"

int main(){
	int test[52];
	int testHost[52];
	int * testDevice;
	hiprandState* devStates;
	hipMalloc(&devStates, sizeof(hiprandState) );
	hipMalloc(&testDevice, sizeof(int)*52);
	testRand<<<1, 1>>>(testDevice, devStates);
	hipMemcpy(testHost, testDevice, sizeof(testDevice),hipMemcpyDeviceToHost);
	for (int i = 0; i < 52; i++){
		std::cout << test[i] << std::endl;
	}
	std::cout << "Press any key to exit";
	std::cin >> new char;
}
