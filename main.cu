#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <vector>
#include <iostream>
#include <fstream>
#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include "berlin52.h"
#include "rand.h"

#define BLOCK_SIZE 32
#define GRID_SIZE 10
#define POPULATION_MULTIPLIER 5

int main(){
	const int gridSize = 10;
	const int populationSize = BLOCK_SIZE*GRID_SIZE;
	const int chromosomeSize = 52;
	int* source;
	int* devicePopulation;
	int* deviceTSPRoute;
	int* hostPopulation;

	hipHostMalloc((void**) &hostPopulation, populationSize*sizeof(int));
	hipMalloc((void**) &source, chromosomeSize*sizeof(int));
	hipMalloc((void**) &devicePopulation, populationSize*sizeof(int));
	hipMalloc((void**) &deviceTSPRoute, chromosomeSize*2*sizeof(float));
	hipMemcpy(deviceTSPRoute, berlin52, chromosomeSize*2*sizeof(float), hipMemcpyHostToDevice);
	thrust::device_ptr<int> sourceThrust = thrust::device_pointer_cast(source);
	thrust::sequence(sourceThrust, sourceThrust + chromosomeSize);

	createRandomPermutation<<<gridSize, BLOCK_SIZE>>>(source, devicePopulation, chromosomeSize, time(NULL));
	hipDeviceSynchronize();

	hipMemcpy(hostPopulation, devicePopulation, populationSize*sizeof(int),hipMemcpyDeviceToHost);
	for (int i = 0; i < populationSize; i++){
		for(int j = 0; j < chromosomeSize; j++){
			std::cout << hostPopulation[j+i*chromosomeSize] << " ";
		}
		std::cout << std::endl;
	}

	hipFree(devicePopulation);
	hipFree(deviceTSPRoute);
	hipFree(source);
	cudaHostFree(hostPopulation);
	std::cin.get();
}