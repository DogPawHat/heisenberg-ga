#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include "global_structs.h"
#include "berlin52.h"

__global__ void createRandomPermutation(deviceFields fields, long seed);
__global__ void runGeneticAlgorithm(deviceFields fields);
__global__ void createRandomSeeds(deviceFields fields, long seed);

int main(){
	deviceFields device;
	hostFields host;

	hipMalloc((void**) &device.population, TOTAL_POPULATION_MEMORY_SIZE*sizeof(short));
	hipMalloc((void**) &device.seeds, POPULATION_SIZE*sizeof(int));
	hipMalloc((void**) &device.TSPGraph, 2*CHROMOSOME_SIZE*sizeof(float));
	hipMalloc((void**) &device.source, CHROMOSOME_SIZE*sizeof(short));

	hipMemcpy(device.TSPGraph, berlin52, 2*CHROMOSOME_SIZE*sizeof(float), hipMemcpyHostToDevice);
	
	thrust::device_ptr<short> sourceThrust(device.source);
	thrust::sequence(sourceThrust, sourceThrust+CHROMOSOME_SIZE);

	createRandomSeeds<<<GRID_SIZE, BLOCK_SIZE>>>(device, time(NULL));
	createRandomPermutation<<<GRID_SIZE, BLOCK_SIZE>>>(device, time(NULL));
	hipDeviceSynchronize();

	runGeneticAlgorithm<<<GRID_SIZE, BLOCK_SIZE>>>(device);
	hipDeviceSynchronize();

	hipMemcpy(host.population, device.population, TOTAL_POPULATION_MEMORY_SIZE*sizeof(short),hipMemcpyDeviceToHost);

	for (short i = 0; i < POPULATION_SIZE; i++){
		for(short j = 0; j < CHROMOSOME_SIZE; j++){
			std::cout << host.population[j+i*CHROMOSOME_SIZE] << " ";
		}
		std::cout << std::endl;
	}

	hipFree(device.population);
	hipFree(device.population);

	std::cin.get();
}