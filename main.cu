#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include "global_structs.h"
#include "berlin52.h"

__global__ void createRandomPermutation(deviceFields*);
__global__ void createRandomSeeds(deviceFields*, long seed);
__global__ void runGeneticAlgorithm(deviceFields*);

void check(hipError_t call){
	if(call != hipSuccess){
		throw &call;
	}
}

int chromosomeCheck(short chromosome[]){
	int k;
	for(int i = 0; i < CHROMOSOME_SIZE; i++){
		k = 0;
		for(int j = 0; j < CHROMOSOME_SIZE; j++){
			if(chromosome[j] == i){
				k++;
			}else if(chromosome[j] > CHROMOSOME_SIZE || chromosome[j] < 0){
				return 1;
			}
		}
		if(k != 1){
			return 1;
		}
	}
	return 0;
}

int main(){
	try{
		deviceFields * device;
		hostFields host;

		check(hipMalloc((void**)&device, sizeof(deviceFields)));

		hipMemcpy(device->TSPGraph, berlin52, 2*CHROMOSOME_SIZE*sizeof(float), hipMemcpyHostToDevice);

		for(int i = 0; i < CHROMOSOME_SIZE; i++){
			host.source[i] = i;
		}

		check(hipMemcpy(device->source, host.source, CHROMOSOME_SIZE*sizeof(short), hipMemcpyHostToDevice));

		createRandomSeeds<<<GRID_SIZE, BLOCK_SIZE>>>(device, time(NULL));
		createRandomPermutation<<<GRID_SIZE, BLOCK_SIZE>>>(device);
		check(hipDeviceSynchronize());

		runGeneticAlgorithm<<<GRID_SIZE, BLOCK_SIZE>>>(device);
		check(hipDeviceSynchronize());

		check(hipMemcpy(host.population, device->population, POPULATION_SIZE*sizeof(metaChromosome),hipMemcpyDeviceToHost));

		for (short i = 0; i < POPULATION_SIZE; i++){
			std::cout << '[' << chromosomeCheck(host.population[i].chromosome) << ']' << " ";
			for(short j = 0; j < CHROMOSOME_SIZE; j++){
				std::cout << host.population[i].chromosome[j] << " ";
			}
			std::cout << host.population[i].distance << " " << host.population[i].fitness << std::endl;
		}
	}catch(hipError_t * e){
		std::cout << "Oh crap: " << hipGetErrorString(*e) << std::endl;
	}
}
