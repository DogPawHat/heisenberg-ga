#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <vector>
#include <iostream>
#include <fstream>
#include "rand.h"



int main(){
	const int testSize = 52;
	float testHost[testSize];
	float * testDevice;
	hipMalloc((void**)&testDevice, testSize*sizeof(float));
	testRand<<<1, 1>>>(testDevice, testSize);
	hipMemcpy(testHost, testDevice, sizeof(float)*testSize,hipMemcpyDeviceToHost);
	for (int i = 0; i < 52; i++){
		std::cout << testHost[i] << std::endl;
	}
	hipFree(testDevice);
	std::cin.get();
}
