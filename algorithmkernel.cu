#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include "ga_struts.h"

__device__ float * selection(int * matingPool, int * islandPoplulation, deviceFields fields, fieldSizes sizes){
	__shared__ float fitnessValues[sizes.populationSize];
	__shared__ float totalFinessValue;
	float selectedChromosome[sizes.chromosomeSize];
	int start = islandPoplulation[threadIdx.x*sizes.chromosoneSize];
	for(int i = 1; i < sizes.chromosoneSize; i++){
		float xd = tspGraph[start+(i*2)] - tspGraph[start+(i*2)-2];
		float yd = tspGraph[start+(i*2)+1] - tspGraph[start+(i*2)-1];
		fitnessValues[threadIdx.x] += sqrt(xd^2 + yd^2);
		totalFitnessValue += sqrt(xd^2 + yd^2);
		__syncthreads();
	}
	fitnessValues[threadIdx.x] = finessValues[threadIdx.x]/totalFitnessValue;
	__syncthreads();

	float rouletteBall = randomrouletteBall();
	unsigned float diff = fdif(fitnessValues[threadIdx.x], rouletteBall);
	memcpy(selectedIndividual, &islandPopulation[threadIdx.x*chromosomeSize], sizeof(int)*chromosomeSize);

	for(int i = 0; i < sizes.poplulationSize; i++){
		if(diff < fitnessValues[i] - rouletteBall){
			fdif(fitnessValues[threadIdx.x], rouletteBall);
			memcpy(selectedChromosome, &islandPopulation[i*chromosomeSize], sizeof(int)*chromosomeSize);
		}
		__syncthreads();
	}
	return selectedIndividual;
}

__device__ void generation(int * islandPopulation, deviceFields fields, fieldSizes sizes){
	__shared__ float matingPool[sizes.populationSize/10];
	memcpy(&matingPool[threadIdx.x*chromosoneSize], selection(matingPool, islandPopulation, fields, sizes), sizeof(float)*chromosomeSize);
	__syncthreads();
	for(int i=0; i < popMultiplier; i++){
		memcpy(&islandPopulation[(threadIdx*popMultiplier+i)*chromosomeSize], &matingPool[threadIdx.x*chromosoneSize], chromosomesize*sizeof(int));
	}
}


__global__ void runGeneticAlgorithm(deviceFields fields, fieldSizes sizes){
	int gridIndex = threadIdx.x + blockDim.x*blockIdx.x;
	__shared__ int * islandPopulation = (int*) malloc(sizes.populationSizes)*sizes.chromosoneSize)];
	memcpy(&islandPopulation[threadIdx*chromosomeSize], &poplulation[gridIndex*chromosomeSize], chromosoneSize*sizeof(int));
	generation(islandPopulation, fields, sizes);
	memcpy(&poplulation[gridIndex*chromosomeSize], &islandPoplulation[blockDim.x*chromosomeSize], chromosomeSize*sizeof(int));
}
