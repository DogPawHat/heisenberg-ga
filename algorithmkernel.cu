#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/random/uniform_real_distribution.h>
#include <thrust/random/uniform_int_distribution.h>
#include "global_structs.h"


__device__ __forceinline__ void crossover(metaChromosome[], deviceFields);
__device__ __forceinline__ void selection(metaChromosome[], deviceFields);
__device__ __forceinline__ void mutation(metaChromosome[], deviceFields);
__device__ __forceinline__ void createNewSeed(deviceFields, long);
__device__ __forceinline__ void distanceEvauation(metaChromosome[], deviceFields);
__device__ __forceinline__ void fitnessEvauation(metaChromosome[], deviceFields);
__device__ __forceinline__ void bitonicSort(metaChromosome islandPopulation[]);


__device__ __forceinline__ void generation(metaChromosome islandPopulation[], deviceFields fields){
	
	if(blockIdx.x <= 8){
		crossover(islandPopulation, fields);
	}else if(blockIdx.x == 9){
		mutation(islandPopulation, fields);
	}
	__syncthreads();

	selection(islandPopulation, fields);
	__syncthreads();

	bitonicSort(islandPopulation);


	if(blockIdx.x <= (BLOCK_SIZE - 2) && threadIdx.x >= BLOCK_SIZE/2){
		fields.population[threadIdx.x*2 + blockDim.x*blockIdx.x +ISLAND_POPULATION_SIZE] = islandPopulation[threadIdx.x];
	}else if(blockIdx.x == (BLOCK_SIZE - 1) && threadIdx.x >= BLOCK_SIZE/2){
		fields.population[threadIdx.x*2] = islandPopulation[threadIdx.x];
	}else if(threadIdx.x <= BLOCK_SIZE/2){
		fields.population[threadIdx.x*2 + blockDim.x*blockIdx.x - 1] = islandPopulation[threadIdx.x];
	}
	__syncthreads();

}


__global__ void runGeneticAlgorithm(deviceFields fields){
	int gridIndex = threadIdx.x + blockDim.x*blockIdx.x;
	__shared__ metaChromosome islandPopulation[ISLAND_POPULATION_SIZE];

	islandPopulation[threadIdx.x] = fields.population[gridIndex];
	__syncthreads();

	for(int i = 0; i < 20; i++){
		generation(islandPopulation, fields);
		__syncthreads();
	}

	fields.population[gridIndex] = islandPopulation[threadIdx.x];
	__syncthreads();
}



/*Selection Functions*/

__device__ __forceinline__ float randomRouletteBall(deviceFields fields){
	thrust::minstd_rand0 rng(fields.seeds[threadIdx.x + blockDim.x*blockIdx.x]);
	thrust::uniform_real_distribution<float> dist(0, 1);
	float result = dist(rng);
	return result;
}

__device__ __forceinline__ void selection(metaChromosome islandPopulation[], deviceFields fields){
	__shared__ metaChromosome selectedPopulation[ISLAND_POPULATION_SIZE];
	float rouletteBall;
	float currentFitnessInterval;

	distanceEvauation(islandPopulation, fields);
	fitnessEvauation(islandPopulation, fields);

	rouletteBall = randomRouletteBall(fields);
	currentFitnessInterval = islandPopulation[0].fitness;
	selectedPopulation[threadIdx.x] = islandPopulation[0];

	for(short i = 1; i < ISLAND_POPULATION_SIZE; i++){
		if(rouletteBall > currentFitnessInterval){
			currentFitnessInterval += islandPopulation[i].fitness;
		}else{
			selectedPopulation[threadIdx.x] = islandPopulation[i];
			break;
		}
	}

	islandPopulation[threadIdx.x] = selectedPopulation[threadIdx.x];
}

__device__ __forceinline__ void distanceEvauation(metaChromosome islandPopulation[], deviceFields fields){
	for(short i = 1; i < CHROMOSOME_SIZE; i++){
		short j  = i - 1;
		float xi = fields.TSPGraph[2*islandPopulation[threadIdx.x].chromosome[i]];
		float xj = fields.TSPGraph[2*islandPopulation[threadIdx.x].chromosome[j]];
		float yi = fields.TSPGraph[2*islandPopulation[threadIdx.x].chromosome[i]+1];
		float yj = fields.TSPGraph[2*islandPopulation[threadIdx.x].chromosome[j]+1];
		float xd = fmaxf(xi, xj) - fminf(xi, xj);
		float yd = fmaxf(yi, yj) - fminf(yi, yj);
		islandPopulation[threadIdx.x].distance += sqrtf(xd*xd + yd*yd);
		__syncthreads();
	}
}

__device__ __forceinline__ void fitnessEvauation(metaChromosome islandPopulation[], deviceFields fields){
	__shared__ float sumOfFitnessValues[ISLAND_POPULATION_SIZE];
	islandPopulation[threadIdx.x].fitness = 1/islandPopulation[threadIdx.x].distance;
	sumOfFitnessValues[threadIdx.x] = islandPopulation[threadIdx.x].fitness;
	__syncthreads();

	for(short stride = 1; stride < ISLAND_POPULATION_SIZE; stride *= 2){
		if(threadIdx.x + stride < ISLAND_POPULATION_SIZE){
			sumOfFitnessValues[threadIdx.x] += sumOfFitnessValues[threadIdx.x+stride];
		}
	}

	islandPopulation[threadIdx.x].fitness = islandPopulation[threadIdx.x].fitness/sumOfFitnessValues[0];
}

__device__ __forceinline__ void bitonicStep(int stride, metaChromosome islandPopulation[]){
	metaChromosome temp;
	if((threadIdx.x % stride) >= 0 && threadIdx.x % stride < stride/2){
		if(islandPopulation[threadIdx.x].fitness > islandPopulation[threadIdx.x + (stride/2)].fitness){
			temp = islandPopulation[threadIdx.x];
			islandPopulation[threadIdx.x] = islandPopulation[threadIdx.x + (stride/2)];
			islandPopulation[threadIdx.x + (stride/2)] = temp;
		}
	}
}

__device__ __forceinline__ void bitonicSort(metaChromosome islandPopulation[]){
	for(int i = 2; i < ISLAND_POPULATION_SIZE; i*2){
		for(int j = i; j <= 1; j/2){
			bitonicStep(j, islandPopulation);
		}
	}
}


/*Genetic Operators*/

__device__ __forceinline__ void crossover(metaChromosome islandPopulation[], deviceFields fields){
	/*We need two different paths here beause each thread needs two parents to generate a single offspring.
	The first half of the block will take one parent from the first half of islandPopulation, while the second parent
	will come from the second half. This is reversed for the second half of the block. To reduce warp control divergence,
	block size shoud be a multiple of 2*warp size, 32 being the current value of warps in Fermi and Kepler GPU's*/
	
	short* parent1; //Points to the first element in the chromosome of parent1
	short* parent2;
	short point1;
	short point2;
	metaChromosome offspring = islandPopulation[threadIdx.x];
	thrust::minstd_rand0 rng(fields.seeds[threadIdx.x+blockDim.x*blockIdx.x]);
	thrust::uniform_int_distribution<short> dist1;
	thrust::uniform_int_distribution<short> dist2;

	if(threadIdx.x < (BLOCK_SIZE/2)){
		parent1 = islandPopulation[threadIdx.x].chromosome;
		parent2 = islandPopulation[threadIdx.x+(BLOCK_SIZE/2)].chromosome;
	}else{
		parent1 = islandPopulation[threadIdx.x].chromosome;
		parent2 = islandPopulation[threadIdx.x-(BLOCK_SIZE/2)].chromosome;
	}

	dist1 = thrust::uniform_int_distribution<short>(0, 52);
	point1 = dist1(rng);
	dist2 = thrust::uniform_int_distribution<short>(point1, 52);
	point2 = dist2(rng);

	for(short i = point1; i <= point2; i++){
		offspring.chromosome[i] = parent2[i];
	}

	for(int i = 0; i < point1; i++){
		for(int j = 0; j < (point2 - point1); j++){
			if(parent1[i] == offspring.chromosome[j]){
				offspring.chromosome[i] = parent2[i];
				goto a;
			}
		}
		offspring.chromosome[i] = parent1[i];
		a:
	}
	
	islandPopulation[threadIdx.x] = offspring;
}

__device__ __forceinline__ void mutation(metaChromosome islandPopulation[], deviceFields fields){
	metaChromosome mutant = islandPopulation[threadIdx.x]; 
	thrust::minstd_rand0 rng(fields.seeds[threadIdx.x+blockDim.x*blockIdx.x]);
	thrust::uniform_int_distribution<short> dist1(0, 10);
	thrust::uniform_int_distribution<short> dist2(0, 52);
	short numOfSwaps = dist1(rng);
	short swapPoint1;
	short swapPoint2;
	short temp;

	for(int i = 0; i < numOfSwaps; i++){
		swapPoint1 = dist2(rng);
		swapPoint2 = dist2(rng);
		temp = mutant.chromosome[swapPoint1];
		mutant.chromosome[swapPoint1] = mutant.chromosome[swapPoint2];
		mutant.chromosome[swapPoint2] = temp;
	}

	islandPopulation[threadIdx.x] = mutant;
}

__device__ __forceinline__ void createNewSeed(deviceFields fields, long seed){
	thrust::minstd_rand0 rng(seed);

	thrust::uniform_int_distribution<int> dist(0,RAND_MAX);
	fields.seeds[threadIdx.x + blockDim.x*blockIdx.x]=dist(rng);
}


