#include "rapidxml.hpp"
#include "rapidxml_utils.hpp"
#include "algorithm.cuh"


geneticAlgorithm * hostAlgorithm;
geneticAlgorithm * deviceAlgorithm;


void check(hipError_t call){
	if(call != hipSuccess){
		throw &call;
	}
}

int chromosomeCheck(int chromosome[]){
	int k;
	for(int i = 0; i < hostAlgorithm->CHROMOSOME_SIZE; i++){
		k = 0;
		for(int j = 0; j < hostAlgorithm->CHROMOSOME_SIZE; j++){
			if(chromosome[j] == i){
				k++;
			}else if(chromosome[j] > hostAlgorithm->CHROMOSOME_SIZE || chromosome[j] < 0){
				return 1;
			}
		}
		if(k != 1){
			return 1;
		}
	}
	return 0;
}

void readDataFromXMLInstance(rapidxml::xml_node<>* graph){
	rapidxml::xml_node<>* vertex = graph->first_node("vertex");
	rapidxml::xml_node<>* edge;
	rapidxml::xml_attribute<>* cost;
	for(int i = 0; i < hostAlgorithm->CHROMOSOME_SIZE; i++){
		edge = vertex->first_node("edge");
		for(int j = 0; j < hostAlgorithm->CHROMOSOME_SIZE; j++){
			double* currentCostHost = &(hostAlgorithm->adjacencyMatrix[i*hostAlgorithm->CHROMOSOME_SIZE+j]);

			if(i == j){
				*currentCostHost = 0;
			}else{
				cost = edge->first_attribute("cost");
				*currentCostHost = *(cost->value());
			}
		}
	}
}

void runGeneticAlgorithm(){
	createRandomPermutation
		<<<
			hostAlgorithm->GRID_SIZE,
			hostAlgorithm->BLOCK_SIZE,
			(hostAlgorithm->ISLAND_POPULATION_SIZE*hostAlgorithm->CHROMOSOME_SIZE*sizeof(int))
		>>>
		(deviceAlgorithm);


	createRandomSeeds<<<
			hostAlgorithm->GRID_SIZE,
			hostAlgorithm->BLOCK_SIZE
		>>>(deviceAlgorithm, time(NULL));
	check(hipDeviceSynchronize());


	for(int i = 0; i < hostAlgorithm->GENERATIONS; i++){
		runOneGeneration
			<<<
			hostAlgorithm->GRID_SIZE,
			hostAlgorithm->BLOCK_SIZE,
			(hostAlgorithm->ISLAND_POPULATION_SIZE*hostAlgorithm->CHROMOSOME_SIZE*sizeof(int) + hostAlgorithm->ISLAND_POPULATION_SIZE*sizeof(double))
			>>>
			(deviceAlgorithm);
		check(hipDeviceSynchronize());
	}
}

int main(int argc, char ** argv){
	try{
		char* filename = argv[1];
		int blockSize = atoi(argv[2]);
		int gridSize = atoi(argv[3]);
		int generations = atoi(argv[4]);


		rapidxml::xml_document<> doc;
		rapidxml::file<> file(filename);
		doc.parse<0>(file.data());
		rapidxml::xml_node<>* graph = doc.first_node("travellingSalesmanProblemInstance")->first_node("graph");


		hostAlgorithm = new geneticAlgorithm(blockSize, gridSize, generations, rapidxml::count_children(graph));
		geneticAlgorithm * hostDevice = new geneticAlgorithm(blockSize, gridSize, generations, rapidxml::count_children(graph));
		check(hipMalloc((void**) &deviceAlgorithm, sizeof(geneticAlgorithm)));

		(hostAlgorithm->seeds) = new int[hostAlgorithm->POPULATION_SIZE];
		hipMalloc((void**) &(hostDevice->seeds), hostAlgorithm->POPULATION_SIZE*sizeof(long));

		(hostAlgorithm->source) = new int[hostAlgorithm->CHROMOSOME_SIZE];
		hipMalloc((void**) &(hostDevice->source), hostAlgorithm->CHROMOSOME_SIZE*sizeof(int));

		(hostAlgorithm->adjacencyMatrix) = new double[hostAlgorithm->CHROMOSOME_SIZE*hostAlgorithm->CHROMOSOME_SIZE];
		hipMalloc((void**) &(hostDevice->adjacencyMatrix), hostAlgorithm->CHROMOSOME_SIZE*hostAlgorithm->CHROMOSOME_SIZE*sizeof(double));

		(hostAlgorithm->populationChromosome) = new int[hostAlgorithm->POPULATION_SIZE*hostAlgorithm->CHROMOSOME_SIZE];
		hipMalloc((void**) &(hostDevice->populationChromosome), sizeof(int)*hostAlgorithm->POPULATION_SIZE*hostAlgorithm->CHROMOSOME_SIZE);

		(hostAlgorithm->populationDistance) = new double[hostAlgorithm->POPULATION_SIZE];
		hipMalloc((void**) &(hostDevice->populationDistance), sizeof(double)*hostAlgorithm->POPULATION_SIZE);

		readDataFromXMLInstance(graph);
		check(hipMemcpy(hostDevice->adjacencyMatrix, hostAlgorithm->adjacencyMatrix, sizeof(int)*hostAlgorithm->CHROMOSOME_SIZE, hipMemcpyHostToDevice));


		for(int i = 0; i < hostAlgorithm->CHROMOSOME_SIZE; i++){
			hostAlgorithm->source[i] = i;
		}

		check(hipMemcpy(hostDevice->source, hostAlgorithm->source, sizeof(int)*hostAlgorithm->CHROMOSOME_SIZE, hipMemcpyHostToDevice));
		check(hipMemcpy(deviceAlgorithm, hostDevice, sizeof(geneticAlgorithm), hipMemcpyHostToDevice));


		runGeneticAlgorithm();

		check(hipMemcpy(hostDevice->populationDistance, hostAlgorithm->populationDistance, sizeof(int)*hostAlgorithm->CHROMOSOME_SIZE, hipMemcpyDeviceToHost));
		check(hipMemcpy(hostDevice->populationChromosome, hostAlgorithm->populationChromosome, sizeof(int)*hostAlgorithm->POPULATION_SIZE*hostAlgorithm->CHROMOSOME_SIZE, hipMemcpyDeviceToHost));

		for (int i = 0; i < hostAlgorithm->POPULATION_SIZE; i++){
			std::cout << '[' << chromosomeCheck(&(hostAlgorithm->populationChromosome[i*hostAlgorithm->CHROMOSOME_SIZE])) << ']' << " ";
			for(int j = 0; j < hostAlgorithm->CHROMOSOME_SIZE; j++){
				std::cout << hostAlgorithm->populationChromosome[i*hostAlgorithm->CHROMOSOME_SIZE+j] << " ";
			}
			std::cout << hostAlgorithm->populationDistance[i] << std::endl;
		}


		free(hostAlgorithm->seeds);
		free(hostAlgorithm->source);
		free(hostAlgorithm->adjacencyMatrix);
		free(hostAlgorithm->populationChromosome);
		free(hostAlgorithm->populationDistance);
		free(hostAlgorithm);
		hipFree(hostDevice->seeds);
		hipFree(hostDevice->source);
		hipFree(hostDevice->adjacencyMatrix);
		hipFree(hostDevice->populationChromosome);
		hipFree(hostDevice->populationDistance);
		hipFree(hostDevice);
		hipFree(deviceAlgorithm);

	}
	catch(hipError_t * e){
		std::cout << "Oh crap: " << hipGetErrorString(*e) << std::endl;
	}
}



